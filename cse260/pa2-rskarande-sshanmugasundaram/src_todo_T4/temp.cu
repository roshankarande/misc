
#include <hip/hip_runtime.h>


/*
    extern __shared__ _FTYPE_ As[];
    //extern __shared__ _FTYPE_ Bs[];

    int block_x = blockIdx.x; int block_y = blockIdx.y;
    int thread_x = threadIdx.x; int thread_y = threadIdx.y;
    int block_dim_x = blockDim.x; int block_dim_y = blockDim.y;
    int grid_dim_x = gridDim.x; int grid_dim_y = gridDim.y;

    int I =  block_y * block_dim_y + thread_y;
    int J =  block_x * block_dim_x + thread_x;

    //printf("block_x %d thread_x %d block_y %d thread_y %d I %d J %d \n",block_x,thread_x,block_y,thread_y,I,J);
    //printf("Thread Dim x %d y %d Block Dim %d %d \n",block_dim_x,block_dim_y,block_dim_x,block_dim_y);
    int npad = ceil(N/((float) TILEDIM_K * 2)) * TILEDIM_K * 2;
    if((I < (npad/2)) && (J < (npad/2))){
        _FTYPE_ _c_n0 = 0;
        _FTYPE_ _c_n1 = 0;
        _FTYPE_ _c_m0 = 0;
        _FTYPE_ _c_m1 = 0;
        for(unsigned int c_tile = 0; c_tile < npad/TILEDIM_K; c_tile++){
            if(I < N && (c_tile*TILEDIM_K + thread_x) < N)
                As[thread_y*block_dim_y+thread_x] = A[I*N + c_tile*TILEDIM_K + thread_x];
            else
                As[thread_y*block_dim_y+thread_x] = 0.0;
            if((I+(npad/2) < N) && (c_tile*TILEDIM_K + thread_x < N))
                As[block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x] = A[(I+(npad/2))*N + c_tile*TILEDIM_K + thread_x];
            else
                As[block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x] = 0.0;
            if((c_tile*TILEDIM_K + thread_y) < N && J < N)
                As[2*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x] = B[(c_tile*TILEDIM_K + thread_y)*N + J];
            else
                As[2*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x] = 0;
            if(((c_tile*TILEDIM_K + thread_y) < N) && (J + (npad/2) < N))
                As[3*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x] = B[(c_tile*TILEDIM_K + thread_y)*N + J + (npad/2)];
            else
                As[3*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x] = 0.0;
            __syncthreads();
            
            //printf("before block_x %d block_y %d thread_x %d thread_y %d I %d J %d c_tile %d As [%d] %f [%d] %f [%d] %f [%d] %f A [%d,%d] %f [%d,%d] %f B [%d,%d] %f [%d,%d] %f \n",block_x,block_y,thread_x,thread_y,I,J,c_tile,thread_y*block_dim_y+thread_x,As[thread_y*block_dim_y+thread_x],block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x,As[block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x],2*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x,As[2*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x],3*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x,As[3*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x],I,c_tile*TILEDIM_K + thread_x,A[I*N + c_tile*TILEDIM_K + thread_x],I+(npad/2),c_tile*TILEDIM_K + thread_x,A[(I+(npad/2))*N + c_tile*TILEDIM_K + thread_x],(c_tile*TILEDIM_K + thread_y), J,B[(c_tile*TILEDIM_K + thread_y)*N + J],(c_tile*TILEDIM_K + thread_y), J + (npad/2),B[(c_tile*TILEDIM_K + thread_y)*N + J + (npad/2)]);

            for (unsigned int k = 0; k < TILEDIM_K; k++) {
                _c_n0 += As[thread_y*block_dim_y+k] * As[2*block_dim_x*block_dim_y + k*block_dim_y+thread_x];
                _c_n1 += As[thread_y*block_dim_y+k] * As[3*block_dim_x*block_dim_y + k*block_dim_y+thread_x];
                _c_m0 += As[block_dim_x*block_dim_y + thread_y*block_dim_y+k] * As[2*block_dim_x*block_dim_y + k*block_dim_y+thread_x];
                _c_m1 += As[block_dim_x*block_dim_y + thread_y*block_dim_y+k] * As[3*block_dim_x*block_dim_y + k*block_dim_y+thread_x];
                //printf("block_x %d block_y %d thread_x %d thread_y %d I %d J %d c_tile %d k %d As [%d] %f [%d] %f Bs [%d] %f [%d] %f c_n0 %f c_n1 %f c_m0 %f c_m1 %f \n",block_x,block_y,thread_x,thread_y,I,J,c_tile,k,thread_y*block_dim_y+k,As[thread_y*block_dim_y+k],block_dim_x*block_dim_y + thread_y*block_dim_y+k,As[block_dim_x*block_dim_y + thread_y*block_dim_y+k],2*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x,As[2*block_dim_x*block_dim_y + thread_y*block_dim_y+thread_x],3*block_dim_x*block_dim_y + k*block_dim_y+thread_x,As[3*block_dim_x*block_dim_y + k*block_dim_y+thread_x],_c_n0,_c_n1,_c_m0,_c_m1);
            }
            __syncthreads();
            //printf("block_x %d block_y %d thread_x %d thread_y %d I %d J %d c_tile %d _c_n0 %f _c_n1 %f \n",block_x,block_y,thread_x,thread_y,I,J,c_tile, _c_n0,_c_n1);
        }
        //if((block_x == (grid_dim_x-1)) && (block_y == (grid_dim_y-1))){
            C[I * N + J] = _c_n0;
            if((I < N) && (J+(npad/2)) < N)
                C[I * N + (J+(npad/2))] = _c_n1;
            if(((I+(npad/2)) < N) && (J < N))
                C[(I + (npad/2)) * N + J] = _c_m0;
            if(((I+(npad/2)) < N) && ((J+(npad/2)) < N))
                C[(I + (npad/2)) * N + (J+(npad/2))] = _c_m1;
            //printf("Result \n");
            //printf("Result block_x %d thread_x %d block_y %d thread_y %d C[%d,%d] %f %f C[%d,%d] %f %f C[%d,%d] %f %f C[%d,%d] %f %f\n",block_x,thread_x,block_y,thread_y, I, J,C[I * N + J],_c_n0,I, (J+(N/2)),C[I * N + (J+(npad/2))],_c_n1,(I + (npad/2)), J,C[(I + (npad/2)) * N + J],_c_m0,(I + (npad/2)),(J+(npad/2)),C[(I + (npad/2)) * N + (J+(npad/2))],_c_m1);
        /*}
        else {
            C[I * N + J] = _c_n0;
            C[I * N + (J+(npad/2))] = _c_n1;
            C[(I + (npad/2)) * N + J] = _c_m0;
            C[(I + (npad/2)) * N + (J+(npad/2))] = _c_m1;
            //printf("Result block_x %d thread_x %d block_y %d thread_y %d C[%d,%d] %f %f C[%d,%d] %f %f C[%d,%d] %f %f C[%d,%d] %f %f\n",block_x,thread_x,block_y,thread_y, I, J,C[I * N + J],_c_n0,I, (J+(N/2)),C[I * N + (J+(npad/2))],_c_n1,(I + (npad/2)), J,C[(I + (npad/2)) * N + J],_c_m0,(I + (npad/2)),(J+(npad/2)),C[(I + (npad/2)) * N + (J+(npad/2))],_c_m1);
        }
    //printf("\n");
*/